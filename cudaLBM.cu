#include "hip/hip_runtime.h"
// http://www.caam.rice.edu/~timwar/CAAM210/Flows.html

#include <math.h>
#include <stdlib.h>
#include <stdio.h>
extern "C"
{
#include "png_util.h"
}

#define FLUID 0
#define WALL 1
#define NSPECIES 9

#define dfloat float

#include "hip/hip_runtime.h"

// loop up 1D array index from 2D node coordinates
__host__ __device__ int idx(int N, int n, int m){
  return n + m*(N+2);
}

void lbmInput(const char *imageFileName,
	      dfloat threshold,
	      int *outN,
	      int *outM,
	      unsigned char **rgb,
	      unsigned char **alpha,
	      int **nodeType){

  int n,m, N,M;

  // read png file
  read_png(imageFileName, &N, &M, rgb, alpha);

  // pad to guarantee space around obstacle and extend the wake
  int Npad = 4*N;
  int Mpad = 2*M;
  
  // threshold walls based on gray scale
  *nodeType = (int*) calloc((Npad+2)*(Mpad+2), sizeof(int));

  // mark pixels by gray scale intensity
  unsigned char *rgbPad   = (unsigned char*) calloc(3*(Npad+2)*(Mpad+2), sizeof(unsigned char));
  unsigned char *alphaPad = (unsigned char*) calloc((Npad+2)*(Mpad+2),   sizeof(unsigned char));
  int wallCount = 0;
  for(m=1;m<=M;++m){
    for(n=1;n<=N;++n){
      int offset = ((n-1)+(m-1)*N);
      dfloat r = (*rgb)[3*offset+0];
      dfloat g = (*rgb)[3*offset+1];
      dfloat b = (*rgb)[3*offset+2];

      // center image in padded region (including halo zone)
      int id = idx(Npad,n+(N/4),m+(M/2));
      (*nodeType)[id] = WALL*(sqrt(r*r+g*g+b*b)<threshold);

      wallCount += (*nodeType)[id];
      rgbPad[3*id+0] = r;
      rgbPad[3*id+1] = g;
      rgbPad[3*id+2] = b;
      alphaPad[id] = 255;
    }
  }

  for(n=1;n<=Npad;++n){
    (*nodeType)[idx(Npad,n,1)] = WALL;
    (*nodeType)[idx(Npad,n,Mpad)] = WALL;
  }
  
  free(*rgb); free(*alpha);
  *rgb = rgbPad;
  *alpha = alphaPad;

  
  
  printf("wallCount = %d (%g percent of %d x %d nodes)\n", wallCount, 100.*((dfloat)wallCount/((Npad+2)*(Mpad+2))), Npad, Mpad);
  
  *outN = Npad;
  *outM = Mpad;
}

  
void lbmOutput(const char *fname,
	       const int *nodeType,
	       unsigned char *rgb,
	       unsigned char *alpha,
	       int N,
	       int M,
	       const dfloat *f){
  int n,m,s;
  FILE *bah = fopen(fname, "w");

  dfloat plotMin = .95, plotMax = 1.05;
  for(m=0;m<=M+1;++m){
    for(n=0;n<=N+1;++n){
      int id = idx(N,n,m);

      // over write pixels in fluid region
      if(nodeType[id]==FLUID){
	unsigned char r,g,b,a;

	// reconstruct macroscopic density
	dfloat rho = 0;
	for(s=0;s<NSPECIES;++s)
	  rho += f[id+s*(N+2)*(M+2)];
	rho = ((rho-plotMin)/(plotMax-plotMin)); // rescale
	r = 0;
	g = 255*(1.-rho);
	b = 255*rho;
	a = 255;

	rgb[idx(N,n,m)*3+0] = r;
	rgb[idx(N,n,m)*3+1] = g;
	rgb[idx(N,n,m)*3+2] = b;
	alpha[idx(N,n,m)] = a;
      }
    }
  }
  
  write_png(bah, N+2, M+2, rgb, alpha);

  fclose(bah);
}

__host__ __device__ void lbmEquilibrium(const dfloat c,
					const dfloat rho,
					const dfloat Ux, 
					const dfloat Uy, 
					dfloat *  feq){

  // resolve macroscopic velocity into lattice particle velocity directions
  const dfloat U2 = Ux*Ux+Uy*Uy;
  
  const dfloat v0 = 0;
  const dfloat v1 = +Ux/c;
  const dfloat v2 = +Uy/c;
  const dfloat v3 = -Ux/c;
  const dfloat v4 = -Uy/c;
  const dfloat v5 =  (+Ux+Uy)/c;
  const dfloat v6 =  (-Ux+Uy)/c;
  const dfloat v7 =  (-Ux-Uy)/c;
  const dfloat v8 =  (+Ux-Uy)/c;
  
  // weights used to compute equilibrium distribution (post collision)
  const dfloat w0 = 4.f/9.f, w1 = 1.f/9.f, w2 = 1.f/9.f, w3 =  1.f/9.f;
  const dfloat w4 = 1.f/9.f, w5 = 1.f/36.f, w6 = 1.f/36.f, w7 = 1.f/36.f, w8 = 1.f/36.f;

  // compute LBM post-collisional 
  feq[0] = rho*w0*(1.f + 3.f*v0 + 4.5f*v0*v0 - 1.5f*U2/(c*c));
  feq[1] = rho*w1*(1.f + 3.f*v1 + 4.5f*v1*v1 - 1.5f*U2/(c*c));
  feq[2] = rho*w2*(1.f + 3.f*v2 + 4.5f*v2*v2 - 1.5f*U2/(c*c));
  feq[3] = rho*w3*(1.f + 3.f*v3 + 4.5f*v3*v3 - 1.5f*U2/(c*c));
  feq[4] = rho*w4*(1.f + 3.f*v4 + 4.5f*v4*v4 - 1.5f*U2/(c*c));
  feq[5] = rho*w5*(1.f + 3.f*v5 + 4.5f*v5*v5 - 1.5f*U2/(c*c));
  feq[6] = rho*w6*(1.f + 3.f*v6 + 4.5f*v6*v6 - 1.5f*U2/(c*c));
  feq[7] = rho*w7*(1.f + 3.f*v7 + 4.5f*v7*v7 - 1.5f*U2/(c*c));
  feq[8] = rho*w8*(1.f + 3.f*v8 + 4.5f*v8*v8 - 1.5f*U2/(c*c));
}

#define TX 32
#define TY 32


// perform lattice streaming and collision steps
__global__ void lbmUpdate(const int N,                  // number of nodes in x
			  const int M,                  // number of nodes in y
			  const dfloat c,                // speed of sound
			  const dfloat tau,              // relaxation rate
			  const int    * __restrict__ nodeType,      // (N+2) x (M+2) node types 
			  const dfloat * __restrict__ f,             // (N+2) x (M+2) x 9 fields before streaming and collisions
			  dfloat * __restrict__ fnew){               // (N+2) x (M+2) x 9 fields after streaming and collisions
  
  // number of nodes in whole array including halo
  int Nall = (N+2)*(M+2);
  
  // physics paramaters
  dfloat tauinv = 1.f/tau;

  // loop over all non-halo nodes in lattice
  int n = 1 + threadIdx.x + blockIdx.x*TX;
  int m = 1 + threadIdx.y + blockIdx.y*TY;

  if(m<M+1 && n<=N+1){
    
    // discover type of node (WALL or FLUID)
    const int nt = nodeType[idx(N,n,m)];
    dfloat fnm[NSPECIES];
    
    // OUTFLOW
    if(n==N+1){
      fnm[0] = f[idx(N,n,  m)   + 0*Nall]; // stationary 
      fnm[1] = f[idx(N,n-1,m)   + 1*Nall]; // E bound from W
      fnm[2] = f[idx(N,n,m-1)   + 2*Nall]; // N bound from S
      fnm[3] = f[idx(N,n,m)     + 3*Nall]; // W bound from E
      fnm[4] = f[idx(N,n,m+1)   + 4*Nall]; // S bound from N
      fnm[5] = f[idx(N,n-1,m-1) + 5*Nall]; // NE bound from SW
      fnm[6] = f[idx(N,n,m-1)   + 6*Nall]; // NW bound from SE
      fnm[7] = f[idx(N,n,m+1)   + 7*Nall]; // SW bound from NE
      fnm[8] = f[idx(N,n-1,m+1) + 8*Nall]; // SE bound from NW
      
    }
    else if(nt == FLUID){
      fnm[0] = f[idx(N,n,  m)   + 0*Nall]; // stationary 
      fnm[1] = f[idx(N,n-1,m)   + 1*Nall]; // E bound from W
      fnm[2] = f[idx(N,n,m-1)   + 2*Nall]; // N bound from S
      fnm[3] = f[idx(N,n+1,m)   + 3*Nall]; // W bound from E
      fnm[4] = f[idx(N,n,m+1)   + 4*Nall]; // S bound from N
      fnm[5] = f[idx(N,n-1,m-1) + 5*Nall]; // NE bound from SW
      fnm[6] = f[idx(N,n+1,m-1) + 6*Nall]; // NW bound from SE
      fnm[7] = f[idx(N,n+1,m+1) + 7*Nall]; // SW bound from NE
      fnm[8] = f[idx(N,n-1,m+1) + 8*Nall]; // SE bound from NW
    }
    else{
      // WALL reflects particles
      fnm[0] = f[idx(N,n,m) + 0*Nall]; // stationary 
      fnm[1] = f[idx(N,n,m) + 3*Nall]; // E bound from W
      fnm[2] = f[idx(N,n,m) + 4*Nall]; // N bound from S
      fnm[3] = f[idx(N,n,m) + 1*Nall]; // W bound from E
      fnm[4] = f[idx(N,n,m) + 2*Nall]; // S bound from N
      fnm[5] = f[idx(N,n,m) + 7*Nall]; // NE bound from SW
      fnm[6] = f[idx(N,n,m) + 8*Nall]; // NW bound from SE
      fnm[7] = f[idx(N,n,m) + 5*Nall]; // SW bound from NE
      fnm[8] = f[idx(N,n,m) + 6*Nall]; // SE bound from NW
    }
    
    // macroscopic density
    const dfloat rho = fnm[0]+fnm[1]+fnm[2]+fnm[3]+fnm[4]+fnm[5]+fnm[6]+fnm[7]+fnm[8];
    
    //    if(rho<1e-4){ printf("rho(%d,%d)=%g\n", n,m,rho); exit(-1); }
    
    // macroscopic momentum
    const dfloat delta2 = 1e-5;
    const dfloat Ux = (fnm[1] - fnm[3] + fnm[5] - fnm[6] - fnm[7] + fnm[8])*c/sqrt(rho*rho+delta2);
    const dfloat Uy = (fnm[2] - fnm[4] + fnm[5] + fnm[6] - fnm[7] - fnm[8])*c/sqrt(rho*rho+delta2);
    
    // compute equilibrium distribution
    dfloat feq[NSPECIES];
    lbmEquilibrium(c, rho, Ux, Uy, feq);
    
    // post collision densities
    fnm[0] -= tauinv*(fnm[0]-feq[0]);
    fnm[1] -= tauinv*(fnm[1]-feq[1]);
    fnm[2] -= tauinv*(fnm[2]-feq[2]);
    fnm[3] -= tauinv*(fnm[3]-feq[3]);
    fnm[4] -= tauinv*(fnm[4]-feq[4]);
    fnm[5] -= tauinv*(fnm[5]-feq[5]);
    fnm[6] -= tauinv*(fnm[6]-feq[6]);
    fnm[7] -= tauinv*(fnm[7]-feq[7]);
    fnm[8] -= tauinv*(fnm[8]-feq[8]);
    
    // store new densities
    const int base = idx(N,n,m);
    fnew[base+0*Nall] = fnm[0];
    fnew[base+1*Nall] = fnm[1];
    fnew[base+2*Nall] = fnm[2];
    fnew[base+3*Nall] = fnm[3];
    fnew[base+4*Nall] = fnm[4];
    fnew[base+5*Nall] = fnm[5];
    fnew[base+6*Nall] = fnm[6];
    fnew[base+7*Nall] = fnm[7];
    fnew[base+8*Nall] = fnm[8];
  }
  
}

void lbmCheck(int N, int M, dfloat *f){

  int n,m,s;
  int nanCount = 0;
  for(s=0;s<NSPECIES;++s){
    for(m=0;m<=M+1;++m){
      for(n=0;n<=N+1;++n){
	nanCount += isnan(f[idx(N,n,m)+s*(N+2)*(M+2)]);
      }
    }
  }
  
  if(nanCount){   printf("found %d nans\n", nanCount); exit(-1); }
}



// set initial conditions (use uniform flow f everywhere)
void lbmInitialConditions(dfloat c, int N, int M, int *nodeType, dfloat *f){
  int n,m,s;
  dfloat feqIC[NSPECIES];
  dfloat feqWALL[NSPECIES];
  dfloat rhoIC = 1.;
  dfloat UxIC = 1.;
  dfloat UyIC = 0.;

  lbmEquilibrium(c, rhoIC, UxIC, UyIC, feqIC);
  lbmEquilibrium(c, rhoIC,    0.,  0., feqWALL);

  for(m=0;m<=M+1;++m){
    for(n=0;n<=N+1;++n){
      int base = idx(N, n, m);
      int s;

      if(n==0){
	//      if(nodeType[idx(N,n,m)]==FLUID){
	for(s=0;s<NSPECIES;++s){
	  f[idx(N,n,m)+s*(N+2)*(M+2)] = feqIC[s];
	}
      }
      else{
	for(s=0;s<NSPECIES;++s){
	  f[idx(N,n,m)+s*(N+2)*(M+2)] = feqWALL[s];
	}
      }
#if 0
      printf("[%g,%g,%g,%g] ",
	     f[idx(N,n,m)+0*(N+2)*(M+2)],
	     f[idx(N,n,m)+1*(N+2)*(M+2)],
	     f[idx(N,n,m)+2*(N+2)*(M+2)],
	     f[idx(N,n,m)+3*(N+2)*(M+2)]);
#endif
    }
    //    printf("\n");
  }
}

int main(int argc, char **argv){

  if(argc!=3){
    printf("usage: ./lbm foo.png threshold\n");
    exit(-1);
  }

  int N, M; // size of lattice
  int n,m;

  // read threshold 
  dfloat threshold = atof(argv[2]);
  char *imageFileName = strdup(argv[1]);

  unsigned char *rgb, *alpha;
  int *nodeType;
  lbmInput(imageFileName, threshold, &N, &M, &rgb, &alpha, &nodeType); 
  
  // physical parameters
  dfloat dx = .01;    // lattice node spacings in x
  dfloat dy = .01;
  dfloat dt = dx*.1; // time step (also determines Mach number)
  dfloat c  = dx/dt; // speed of sound
  dfloat tau = .525; // relaxation rate
  dfloat Reynolds = 2./((tau-.5)*c*c*dt/3.);

  printf("Reynolds number %g\n", Reynolds);

  // create lattice storage
  dfloat *f    = (dfloat*) calloc((N+2)*(M+2)*NSPECIES, sizeof(dfloat));
  dfloat *fnew = (dfloat*) calloc((N+2)*(M+2)*NSPECIES, sizeof(dfloat));

  // set initial flow densities
  lbmInitialConditions(c, N, M, nodeType, f);
  lbmInitialConditions(c, N, M, nodeType, fnew);
  
  // DEVICE storage
  dfloat *c_f, *c_fnew;
  int *c_nodeType;
  
  hipMalloc(&c_f, (N+2)*(M+2)*NSPECIES*sizeof(dfloat));
  hipMalloc(&c_fnew, (N+2)*(M+2)*NSPECIES*sizeof(dfloat));
  hipMalloc(&c_nodeType, (N+2)*(M+2)*sizeof(int));

  hipMemcpy(c_f, f, (N+2)*(M+2)*NSPECIES*sizeof(dfloat), hipMemcpyHostToDevice);
  hipMemcpy(c_fnew, fnew, (N+2)*(M+2)*NSPECIES*sizeof(dfloat), hipMemcpyHostToDevice);
  hipMemcpy(c_nodeType, nodeType, (N+2)*(M+2)*sizeof(int), hipMemcpyHostToDevice);
    
  int Nsteps = 60000/2, tstep = 0, iostep = 100;

  // time step
  for(tstep=0;tstep<Nsteps;++tstep){

    // perform two updates
    dim3 T(TX,TY,1);
    dim3 B( (N+1+TX-1)/TX, (M+1+TY-1)/TY, 1);
    
    lbmUpdate <<< B, T >>> (N, M, c, tau, c_nodeType, c_f, c_fnew);
    lbmUpdate <<< B, T >>> (N, M, c, tau, c_nodeType, c_fnew, c_f);

    if(!(tstep%iostep)){ // output an image every iostep
      printf("tstep = %d\n", tstep);
      char fname[BUFSIZ];
      sprintf(fname, "bah%06d.png", tstep);

      hipMemcpy(f, c_f, (N+2)*(M+2)*NSPECIES*sizeof(dfloat), hipMemcpyDeviceToHost);
      lbmOutput(fname, nodeType, rgb, alpha, N, M, f);
    }
  }

  // output final result as image
  hipMemcpy(f, c_f, (N+2)*(M+2)*NSPECIES*sizeof(dfloat), hipMemcpyDeviceToHost);
  lbmOutput("bahFinal.png", nodeType, rgb, alpha, N, M, f);

  exit(0);
  return 0;
}
  
